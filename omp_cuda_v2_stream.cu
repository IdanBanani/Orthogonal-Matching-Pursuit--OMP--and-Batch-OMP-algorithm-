#include "hip/hip_runtime.h"
/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

/* Includes, cuda */
//#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

/* Number of columns & rows in dictionary */
// TODO: get as input
#define M 300  // num of Dictionary columns
#define N 50  // num of Dictionary rows
#define X 25// number of signals
/* Number of non-zero elements in signal */
int K = 4;
/* Residual error */
double epsilon = 1.0e-7;
/* Max num of iterations - assume as same as num of elements in signal */
int T = N;
/* Sign function */
double sign(double x){return (x>=0) - (x<0);}

/* Matrix indexing convention */
#define id(m, n, ld) (((n) * (ld) + (m)))

int main(int argc, char** argv)
{
	hipError_t cudaStat;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	hipStream_t stream[4];
	double *h_D, *h_X, *h_C, *c; //host memory pointers
	double *d_D = 0, *d_S = 0, *d_R = 0; //device memory pointers
	int i;
	int MX = M*X;
	int NX = M*X;
	int MN = M*N, m, n, k, q, t;
	const double alpha = 1.0;
	const double beta = 0.0;
	double norm = sqrt(N), normi, normf, a, dtime;
	printf("\nDictionary dimensions: N x M = %d x %d, K = %d, Number of Signals = %d", N, M, K, X);

	/* Initialize srand and clock */
	srand(time(NULL));
 	clock_t start = clock();


	/* Initialize cublas */
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
    	printf ("CUBLAS initialization failed\n");
    	return EXIT_FAILURE;
    }

    for (i=0 ; i<4 ; i++){
    cudaStat = hipStreamCreate(&stream[i]);
        if (cudaStat != hipSuccess) {
            fprintf (stderr,"! stream create error\n");
            return EXIT_FAILURE;
        }
	hipblasSetStream(handle, stream[i] );

 }
	/* Initialize dictionary on host */
	hipHostMalloc((double*)h_D, (MN * sizeof(h_D[0]))
	if (cudaStat != hipSuccess) {
            fprintf (stderr,"! stream D error\n");
            return EXIT_FAILURE;
    }
	
	for(n = 0; n < N; n++){
		for(m = 0; m < M; m++){	
		a = sign(2.0*rand()/(double)RAND_MAX-1.0)/norm;
		h_D[id(m, n, M)] = a;
	 }
	}
	
	/* Create X random K-sparse signals */
	hipHostMalloc((double*)h_X, (M*X * sizeof(h_X[0]))
	if (cudaStat != hipSuccess) {
            fprintf (stderr,"! stream X error\n");
            return EXIT_FAILURE;
    }
	
	for (i = 0;i < X;i++){
		for(k = 0; k < K; k++){
		a = 2.0*rand()/(double)RAND_MAX - 1.0;
		h_X[(rand()%M)+i*M] = a;}
	}

	/* Allocate solution memory on host */
	hipHostMalloc((double*)h_C, (M*X * sizeof(h_C[0]))
	if (cudaStat != hipSuccess) {
            fprintf (stderr,"! stream X error\n");
            return EXIT_FAILURE;
    }
	
	
	c = (double*)calloc(1, sizeof(c));
	if(c == 0){
		fprintf(stderr, " host memory allocation error (c)\n");
		return EXIT_FAILURE;
	}
	
	



	/* Host to device data transfer: dictionary */
    cudaStat = hipMalloc ((void**)&d_D, (MN)*sizeof(d_D[0]));
    if (cudaStat != hipSuccess) {
        fprintf (stderr,"! device memory allocation error (dictionary)\n");
        return EXIT_FAILURE;
    }

	
	//trasnfer the Host dictionary to Device dictionary
	status = hipblasSetVector(MN, sizeof(h_D[0]),h_D, 1, d_D, 1);
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! device access error (write dictionary)\n");
		return EXIT_FAILURE;
	}
	
	/* Host to device data transfer: signal */
    cudaStat = hipMalloc ((void**)&d_R, MX*sizeof(d_R[0]));
    if (cudaStat != hipSuccess) {
        fprintf (stderr, "! device memory allocation error (signal)\n");
        return EXIT_FAILURE;
    }

	
	status = hipblasSetVector(MX, sizeof(h_X[0]),h_X, 1, d_R, 1);
	if(status != HIPBLAS_STATUS_SUCCESS){
		fprintf(stderr, "! device access error (write signal)\n");
		return EXIT_FAILURE;
	}
	
	/*Allocate device memory for Signal Solution */
    cudaStat = hipMalloc ((void**)&d_S, NX*sizeof(d_S[0]));
    if (cudaStat != hipSuccess) {
        fprintf (stderr, "! device memory allocation error (projected vector)\n");
        return EXIT_FAILURE;
    }


/* Encoding the signal on device*/

	for (i = 0;i<X;i++)	{
		status = hipblasDgemv(handle,HIPBLAS_OP_T, M, N, &alpha, d_D, M,d_R+i*M, 1, &beta, d_S+i*N, 1);
		if(status != HIPBLAS_STATUS_SUCCESS)
		{
			fprintf(stderr, "! kernel execution error (encoding)\n");
			return EXIT_FAILURE;
		}
	}

		//dtime = ((double)clock()-start)/CLOCKS_PER_SEC; // TODO : need to remove
		//printf("\nTime for encoding: %f(s)",dtime);


	/* Decoding the signal on device*/
	start = clock();
	for (i = 0;i<X;i++)	{

		
		//hipStreamSynchronize(stream[i]);

		hipblasDnrm2(handle,N, d_S+i*N, 1,&normi);
		epsilon = sqrt(epsilon*normi);
		normf = normi;
		t = 0;
		while(normf > epsilon && t < T){
			//printf("\n %f",normf);
			hipblasDgemv(handle,HIPBLAS_OP_N, M, N, &alpha, d_D, M,d_S+i*N, 1, &beta, d_R+i*M, 1);
			hipblasIdamax(handle,M, d_R+i*M, 1,&q);
			q = q - 1;
			hipblasGetVectorAsync(1, sizeof(c),&d_R[q+i*M], 1, c, 1);
			h_C[q+i*M] = *c + h_C[q+i*M];
			*c = -(*c);
			hipblasDaxpy (handle,N,c, &d_D[q], M, d_S+i*N, 1);
			hipblasDnrm2(handle,N, d_S+i*N, 1,&normf);
			t++;

		}
		
		/*
		status = cublasGetError();
		if(status != HIPBLAS_STATUS_SUCCESS){
			fprintf(stderr, "! kernel execution error (decoding)\n");
			return EXIT_FAILURE;
		*/

		a = 100.0*(normf*normf)/(normi*normi);
	//	printf("\nComputation residual error: %f",a);

		a=0; q=0; *c=0;
		epsilon=1.0e-7;
	}

	dtime = (((double)clock()-start))/CLOCKS_PER_SEC;
	printf("\n Total time : %f(s) ",dtime);
/* Check the solution */
/*
	printf("\nSolution (first column),Reference (second column):");
	getchar();  // Wait for key ...
	for(m=0; m<M; m++)
	{
		printf("\n%f\t%f\t%f\t%f", h_C[m], h_X[m],h_C[m+M],h_X[m+M]);
	}
	normi = 0; normf = 0;
	for(m=0; m<M; m++)
	{
		normi = normi + h_X[m]*h_X[m];
		normf = normf +
		(h_C[m] - h_X[m])*(h_C[m] - h_X[m]);
	}
	printf("\nSolution residual error:%f", 100.0*normf/normi);
*/
/* Memory clean up */
	for (i=0 ; i<4 ; i++){
	hipStreamDestroy(stream[i]);
	}
	free(h_D);	free(h_X);	free(h_C);
	cudaStat = hipFree(d_D);
	if (cudaStat != hipSuccess) {
		fprintf(stderr,"! device memory free error\n");
	        return EXIT_FAILURE;
	    }
	cudaStat = hipFree(d_S);
	if (cudaStat != hipSuccess) {
		fprintf(stderr,"! device memory free error\n");
	        return EXIT_FAILURE;
	    }
	cudaStat = hipFree(d_R);
	if (cudaStat != hipSuccess) {
		fprintf(stderr,"! device memory free error\n");
	        return EXIT_FAILURE;
	    }

/* Shutdown */


status = hipblasDestroy(handle);
if(status != HIPBLAS_STATUS_SUCCESS){
	fprintf(stderr,"! cublas shutdown error\n");
	return EXIT_FAILURE;
}
	if(argc<=1 || strcmp(argv[1],"-noprompt")){
		printf("\nPress ENTER to exit...\n");
		getchar();
	}
	return EXIT_SUCCESS;
}
